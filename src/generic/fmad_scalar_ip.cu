#include "hip/hip_runtime.h"
/**
 * \file
 * Fill a gpu-based array with a constant value.
 * \todo This only works for contiguous strides.
 */
#include "../core.h"
#include "../ops.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdint.h>
#include <limits>
#include <algorithm>
#include "macros.h"
TYPEDEFS;

#define ENDL "\n"
#define LOG(...) ndLogError(dst,__VA_ARGS__)
#define TRY(e)   do{if(!(e)) {LOG("%s(%d): %s()"ENDL "\tExpression evaluated as failure."ENDL "\t%s"ENDL,__FILE__,__LINE__,__FUNCTION__,#e); goto Error; }}while(0)
#define CUTRY(e) do{hipError_t ecode=(e); if(ecode!=hipSuccess) {LOG("%s(%d): %s()"ENDL "\tExpression evaluated as failure."ENDL "\t%s"ENDL "\t%s"ENDL,__FILE__,__LINE__,__FUNCTION__,#e,hipGetErrorString(ecode)); goto Error; }}while(0)
#define FAIL     LOG("%s(%d) %s()"ENDL "\tExecution should not have reached here."ENDL,__FILE__,__LINE__,__FUNCTION__); goto Error


template<typename T>
__device__ T saturate(float v)
{ return std::max(std::numeric_limits<T>::min(),std::min(v,std::numeric_limits<T>::max()));
}

template<typename T,unsigned BX,unsigned BY,unsigned WORK>
__global__ void __launch_bounds__(BX*BY,1)
fmad_scalar_ip_kernel(T* dst,unsigned w,unsigned h,float m,float b)
{ const int ox=threadIdx.x+(blockIdx.x*WORK)*BX,
            oy=threadIdx.y+ blockIdx.y      *BY;
  if(oy<h)
  { dst+=ox+oy*(int)w;
    if(blockIdx.x!=(gridDim.x-1))
    {
      #pragma unroll
      for(int i=0;i<WORK;++i) dst[i*BX]=m*dst[i*BX]+b;
    } else
    { // last block 
      #pragma unroll
      for(int i=0;i<WORK;++i) if(w-ox>i*BX) dst[i*BX]=m*dst[i*BX]+b;
    }
  } 
}

static unsigned prod(size_t n, size_t *v)
{ size_t o=1;
  while(n-->0) o*=v[n];
  return (unsigned)o;
}

extern "C" unsigned fmad_scalar_ip_cuda(nd_t dst,float m, float b)
{ unsigned w=ndshape(dst)[0],
           h=prod(ndndim(dst)-1,ndshape(dst)+1);
  const unsigned BX=32,BY=32,WORK=8;
  dim3 blocks((unsigned)ceil(w/(float)(WORK*BX)), (unsigned)ceil(h/(float)BY)),
       threads(BX,BY); // run max threads per block (1024).  Set BX to be 1 warp (32).
  /// @cond DEFINES
  #define CASE(T) fmad_scalar_ip_kernel<T,BX,BY,WORK><<<blocks,threads,0,(hipStream_t)ndCudaStream(dst)>>>((T*)nddata(dst),w,h,m,b); break
       {TYPECASE(ndtype(dst));}
  #undef CASE
  /// @endcond
  CUTRY(hipGetLastError());
  return 1;
Error:
  return 0;
}