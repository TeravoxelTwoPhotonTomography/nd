#include "hip/hip_runtime.h"
/**
 * \file
 * Fill a gpu-based array with a constant value.
 * \todo This only works for contiguous strides.
 */
#include "../core.h"
#include "../ops.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdint.h>
#include "macros.h"
TYPEDEFS;

#define ENDL "\n"
#define LOG(...) ndLogError(dst,__VA_ARGS__)
#define TRY(e)   do{if(!(e)) {LOG("%s(%d): %s()"ENDL "\tExpression evaluated as failure."ENDL "\t%s"ENDL,__FILE__,__LINE__,__FUNCTION__,#e); goto Error; }}while(0)
#define CUTRY(e) do{hipError_t ecode=(e); if(ecode!=hipSuccess) {LOG("%s(%d): %s()"ENDL "\tExpression evaluated as failure."ENDL "\t%s"ENDL "\t%s"ENDL,__FILE__,__LINE__,__FUNCTION__,#e,hipGetErrorString(ecode)); goto Error; }}while(0)
#define FAIL     LOG("%s(%d) %s()"ENDL "\tExecution should not have reached here."ENDL,__FILE__,__LINE__,__FUNCTION__); goto Error


template<typename T,unsigned BX,unsigned BY,unsigned WORK>
__global__ void __launch_bounds__(BX,BY)
fill_kernel(T* dst,unsigned w,unsigned h,T v)
{ const int ox=threadIdx.x+(blockIdx.x*WORK)*BX,
            oy=threadIdx.y+ blockIdx.y      *BY;
  if(oy<h)
  { dst+=ox+oy*(int)w;
    if(blockIdx.x!=(gridDim.x-1))
    {
      #pragma unroll
      for(int i=0;i<WORK;++i) dst[i*BX]=v;      
    } else
    { // last block 
      #pragma unroll
      for(int i=0;i<WORK;++i) if(w-ox>i*BX) dst[i*BX]=v;
    }
  }
}

static unsigned prod(size_t n, size_t *v)
{ unsigned o=1;
  while(n-->0) o*=v[n];
  return o;
}

extern "C" unsigned fill_cuda(nd_t dst,uint64_t v)
{ unsigned w=ndshape(dst)[0],
           h=prod(ndndim(dst)-1,ndshape(dst)+1);
  const unsigned BX=32,BY=4,WORK=8;
  dim3 blocks((unsigned)ceil(w/(float)(WORK*BX)), (unsigned)ceil(h/(float)BY)),
       threads(BX,BY);
  /// @cond DEFINES
  #define CASE(T) fill_kernel<T,BX,BY,WORK><<<blocks,threads,0,ndCudaStream(dst)>>>((T*)nddata(dst),w,h,*(T*)&v); break
       {TYPECASE(ndtype(dst));}
  #undef CASE
  /// @endcond
  CUTRY(hipGetLastError());
  return 1;
Error:
  return 0;
}