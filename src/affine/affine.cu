#include "hip/hip_runtime.h"
/**
 * \file
 * nd affine transform on the GPU with CUDA.
 */
#include "../core.h"
#include "../ops.h"

#include "stdio.h"
#include <stdint.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "math_functions.h"


#include "generic/macros.h"
TYPEDEFS;

/// @cond DEFINES
#define MAXDIMS          8  // should be sizeof uchar
#define WARPS_PER_BLOCK  9
#define BLOCKSIZE       (32*WARPS_PER_BLOCK) // threads per block

#define ENDL "\n"
#define LOG(...) ndLogError(dst_,__VA_ARGS__)
#define CUTRY(e) do{hipError_t ecode=(e); if(ecode!=hipSuccess) {LOG("%s(%d): %s()"ENDL "\tExpression evaluated as failure."ENDL "\t%s"ENDL "\t%s"ENDL,__FILE__,__LINE__,__FUNCTION__,#e,hipGetErrorString(ecode)); goto Error; }}while(0)
#define FAIL     LOG("%s(%d) %s()"ENDL "\tExecution should not have reached here."ENDL,__FILE__,__LINE__,__FUNCTION__); goto Error

#define shared extern "C"
#ifndef restrict
#define restrict __restrict__
#endif

// printf() is only supported
// for devices of compute capability 2.0 and higher
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
    #define printf(f, ...) ((void)(f, __VA_ARGS__),0)
#endif
/// @cond DEFINES

typedef uint8_t      u8;
typedef uint32_t     u32;
typedef unsigned int uint;

typedef struct arg_t_
{ u8                 ndim;      ///< The number of dimensions
  u32                nelem;     ///< The total number of elements
  size_t   *restrict shape;     ///< Buffer of length ndim,  ordered [w,h,d,...].  Always agrees with stride.  Maintained for convenience.
  size_t   *restrict strides;   ///< Buffer of length ndim+1, strides[i] is the number of bytes layed out between unit steps along dimension i  
  void     *restrict data;      ///< A poitner to the data.
} arg_t;

inline __device__ float clamp(float f, float a, float b)          {return fmaxf(a, fminf(f, b));}
template<class T> inline __device__ T saturate(float f);
template<> inline __device__ uint8_t  saturate<uint8_t>(float f)  {return clamp(f,0,UCHAR_MAX);}
template<> inline __device__ uint16_t saturate<uint16_t>(float f) {return clamp(f,0,USHRT_MAX);}
template<> inline __device__ uint32_t saturate<uint32_t>(float f) {return clamp(f,0,ULONG_MAX);}
template<> inline __device__ uint64_t saturate<uint64_t>(float f) {return clamp(f,0,ULLONG_MAX);} // FIXME - will overflow float type
template<> inline __device__  int8_t  saturate< int8_t> (float f) {return clamp(f,CHAR_MIN,CHAR_MAX);}
template<> inline __device__  int16_t saturate< int16_t>(float f) {return clamp(f,SHRT_MIN,SHRT_MAX);}
template<> inline __device__  int32_t saturate< int32_t>(float f) {return clamp(f,LONG_MIN,LONG_MAX);}
template<> inline __device__  int64_t saturate< int64_t>(float f) {return clamp(f,LLONG_MIN,LLONG_MAX);} // FIXME - will overflow float type
template<> inline __device__  float   saturate<float>(float f)    {return f;}
template<> inline __device__  double  saturate<double>(float f)   {return f;}

inline __device__ float fpartf(float f) { return f-(long)f;}

inline __device__ u8 inbounds_(float x,size_t n)
{return floorf(x)>=0.0f && floorf(x)<n;}

inline __device__ uchar2 inbounds(u8 ndim,const size_t*restrict const shape, const float*restrict const r)
{ uchar2 b=make_uchar2(0,0);
  for(u8 i=0;i<ndim;++i)
  { b.x|=(inbounds_(r[i]     ,shape[i])<<i);
    b.y|=(inbounds_(r[i]+1.0f,shape[i])<<i);
  }
  return b;
}

/**
 * nD linear interpolation for maximum intensity composting.
 *
 * The boundary handling used here is designed for maximum intensity composting.
 * A constant (determined by \a param->boundary_value) is returned for 
 * out-of-bounds samples.
 * Samples stradling the border are handled as a special case.
 */
template<class Tsrc,class Tdst>
inline __device__ Tdst sample(arg_t &src,const float *restrict const r,const nd_affine_params_t*const param)
{ uchar2 bounds=inbounds(src.ndim,src.shape,r); // bit i set if inbounds on dim i  
  // clamp to boundary value for out-of-bounds
  if(!bounds.x && !bounds.y)
    return param->boundary_value;
  
  // compute offset to top left ish corner of lattice unit
  u32 idx=0;
  for(u8 i=0;i<src.ndim;++i) 
    idx+=src.strides[i]*floorf(r[i]);

  // iterate over each corner of hypercube
  float v(0.0f);
  for(u8 i=0;i<((1<<src.ndim)-1);++i)              // bits of i select left or right sample on each dimension
  { uchar2 o=make_uchar2(~i&~bounds.x,i&~bounds.y);// don't need to mask high bits of i
    float w=1.0f;
    int offset=0; // offset so corner clamps to edge
    for(u8 idim=0;idim<src.ndim;++idim)            // loop for dot-products w bit vector
    { const size_t s=src.strides[idim];
      const float  a=fpartf(r[idim]),
                   b=1.0f-a;
#define BIT(bs_,i_) (((bs_)&(1<<(i_)))!=0)
      offset+=BIT(o.x,idim)*s          // clamp corner (top left ish)
             -BIT(o.y,idim)*s          // clamp corner (bot right ish)
             +BIT(i,idim)  *s;         // normal corner offset
      w*=BIT(i,idim)*a+BIT(~i,idim)*b; // weight for corner is a product of lerp weights for each dimension
#undef BIT
    }
    v+=w*((Tsrc*)src.data)[idx+offset];             // weighted value for corner
  }
  return saturate<Tdst>(v);
}

/**
 * Yield a position vector from an index.
 * For r=(x,y,z...) in a box with dimensions (Nx,Ny,Nz,..)
 * idx = x+Nx(y+Ny*(z+Nz(...)))
 */
inline __device__ void idx2pos(u8 ndim, const size_t *restrict const shape, unsigned idx, unsigned *restrict r)
{ for(u8 i=0;i<ndim;++i)
  { r[i]=idx%shape[i];
    idx/=shape[i];
  }
}

/**
 * Transform input vector according to an affine projection matrix.
 * \verbatim
 *             T
 * [lhs 1] = [m b] * [rhs]
 *           [0 1]   [1  ]
 * \endverbatim
 * 
 * \param[in,out] lhs  Output vector (left-hand side).
 * \param[in]     nlhs Number of elements in \a lhs.
 * \param[in]     T    A (\a nlhs+1)-by-(\a nrhs+1) row-major affine projection matrix.
 * \param[in]     rhs  Input vector (right-hand side).
 * \param[in]     nrhs Number of elements in \a rhs.
 */
inline __device__ void proj(
           float *restrict       lhs,
              u8                 nlhs,
  const   double *restrict const T,
  const unsigned *restrict const rhs,
              u8                 nrhs
  )
{ for(unsigned r=0;r<nlhs;++r)
  { lhs[r]=0.0f;
    for(unsigned c=0;c<nrhs;++c)
      lhs[r]+=rhs[c]*T[(nrhs+1)*r+c];
    lhs[r]+=T[(nrhs+1)*r+nrhs];
  }
}

#define max(a,b) ((a)>(b))?(a):(b)

template<typename Tsrc,typename Tdst> 
__global__ void affine_kernel(arg_t dst, arg_t src, const double *transform, const nd_affine_params_t param)
{ 
  Tdst     obuf=0;
  Tdst     ibuf=0;  
  unsigned rdst[MAXDIMS];
  float    rsrc[MAXDIMS];
  unsigned idst=threadIdx.x+blockIdx.x*blockDim.x;
#if 0
  if(blockIdx.x==0 && threadIdx.x==2)
    printf("ksize src:%d dst:%d\n",(int)sizeof(*ibuf),(int)sizeof(*obuf));
#endif
  if(idst<dst.nelem)
  { idx2pos(dst.ndim,dst.shape,idst,rdst);
    proj(rsrc,src.ndim,transform,rdst,dst.ndim);
    ibuf=sample<Tsrc,Tdst>(src,rsrc,&param);
    obuf=((Tdst*)dst.data)[idst];
    __syncthreads();
//    ((Tdst*)dst.data)[idst]=ibuf;
    if(ibuf>obuf)
      ((Tdst*)dst.data)[idst]=ibuf;
    else
      ((Tdst*)dst.data)[idst]=obuf;
//    ((Tdst*)dst.data)[idst]=max(obuf,ibuf);
  }
}

static arg_t make_arg(const nd_t a)
{ arg_t out = 
  { (u8)     ndndim(a),
    (u32)    ndnelem(a),
    (size_t*)ndCudaShape(a),
    (size_t*)ndCudaStrides(a),
    nddata(a)
  };
  return out;
}

//
// === Interface ===
//



/**
 * Assume the ndkind() of \a src_ and \a dst_ have already been checked.
 */
shared unsigned ndaffine_cuda(nd_t dst_, const nd_t src_, const double *transform, const nd_affine_params_t *param)
{ arg_t dst=make_arg(dst_),
        src=make_arg(src_);
  /// @cond DEFINES
  #define CASE2(TSRC,TDST)  printf("size src:%d dst:%d\n",(int)sizeof(TSRC),(int)sizeof(TDST));affine_kernel<TSRC,TDST><<<1+(unsigned)dst.nelem/BLOCKSIZE,BLOCKSIZE,0,0>>>(dst,src,transform,*param); break  
  #define CASE(T) TYPECASE2(ndtype(dst_),T); break
  /// @endcond
  TYPECASE(ndtype(src_));
  #undef CASE
  #undef CASE2
  CUTRY(hipGetLastError());
  return 1;
Error:
  return 0;
}